#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <sys/time.h>
#include <stdlib.h>
#include <thrust/scan.h>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

using namespace std;

// OUTCORE
__global__ void edge_process_outcore(const edge_node *L, const unsigned int edge_num, unsigned int *distance_prev, unsigned int *distance_cur, int *anyChange, unsigned int *pred){
	
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int thread_num = blockDim.x * gridDim.x;

	int warp_id = thread_id/32;
	int warp_num = thread_num % 32 ? thread_num/32 + 1 : thread_num/32;
	int lane_id = thread_id % 32;

	int load = (edge_num % warp_num == 0) ? edge_num/warp_num : edge_num/warp_num+1;
	int beg = load * warp_id;
	int end = min(edge_num, beg + load);
	beg += lane_id;

	unsigned int u;
	unsigned int v;
	unsigned int w;

	for(int i = beg; i < end; i+=32){
		u = L[i].srcIndex;
		v = L[i].destIndex;
		w = L[i].weight;
		if(distance_prev[u] == UINT_MAX){
			continue;
		} else if(distance_prev[u] + w < distance_cur[v]){
			anyChange[0] = 1;
			pred[v] = 1;
			atomicMin(&distance_cur[v], distance_prev[u] + w);
		}
	}
}

// INCORE 
__global__ void edge_process_incore(const edge_node *L, const unsigned int edge_num, unsigned int *distance, int *anyChange, unsigned int *pred){

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int thread_num = blockDim.x * gridDim.x;

	int warp_id = thread_id/32;
	int warp_num = thread_num % 32 ? thread_num/32 + 1 : thread_num/32;
	int lane_id = thread_id % 32;

	int load = (edge_num % warp_num == 0) ? edge_num/warp_num : edge_num/warp_num+1;
	int beg = load * warp_id;
	int end = min(edge_num, beg + load);
	beg += lane_id;

	unsigned int u;
	unsigned int v;
	unsigned int w;

	for(int i = beg; i < end; i+=32){
		u = L[i].srcIndex;
		v = L[i].destIndex;
		w = L[i].weight;
		int dist = distance[u] + w;
		if(distance[u] == UINT_MAX){
			continue;
		} else if(dist < distance[v]){
			//printf("src is %u , dest is %u, weight is %u\n", u, v, w);

			anyChange[0] = 1;
			pred[v] = 1;
			atomicMin(&distance[v], dist);
		}
	}
}

__global__ void getX(const edge_node *L, const unsigned int edge_num, unsigned int *pred, unsigned int *X){
    
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_num = blockDim.x * gridDim.x;

    int warp_id = thread_id/32;
    int warp_num = (thread_num % 32 == 0) ? thread_num/32 : edge_num/32 + 1;
    int lane_id = thread_id % 32;

    // how many edges each warp takes
    int load = (edge_num % warp_num == 0) ? edge_num/warp_num : edge_num/warp_num+1;
    int beg = load * warp_id;
    int end = min(edge_num, beg+ load);
    beg += lane_id;

    unsigned int num = 0;
    for(int i = beg; i < end; i+=32){
	int mask = __ballot(pred[L[i].srcIndex]);
	if(lane_id == 0){
	    num += (unsigned int) __popc(mask);
	}
    }

    if(lane_id == 0){
	X[warp_id] = num;
    }
}

__global__ void getY(unsigned int *X){
   
    int n = blockDim.x;
    int thid = threadIdx.x;
    int offset = 1;

    for(int d = n >> 1; d > 0; d >>= 1){
	__syncthreads();
	if( thid < d ){
	    int ai = offset*(2*thid+1)-1;
	    int bi = offset*(2*thid+2)-1;
	    X[bi] += X[ai];
	}
	offset *= 2;
    }
    
    if( thid == 0) { X[n-1] = 0; }

    for(int d = 1 ; d < n ; d *= 2){
	offset >>= 1;
	__syncthreads();
	if( thid < d ){
	    int ai = offset*(2*thid+1)-1;
	    int bi = offset*(2*thid+2)-1;
	    int t = X[ai];
	    X[ai] = X[bi];
	    X[bi] += t;
	}
    }
    __syncthreads();
}

__global__ void getT(const edge_node *L, const unsigned int edge_num, unsigned int *pred, unsigned int *Y, edge_node *T){
    // fill here
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int thread_num = blockDim.x * gridDim.x;

    int warp_id = thread_id/32;
    int warp_num = (thread_num % 32 == 0) ? thread_num/32 : edge_num/32 + 1;
    int lane_id = thread_id % 32;

    // how many edges each warp takes
    int load = (edge_num % warp_num == 0) ? edge_num/warp_num : edge_num/warp_num+1;
    int beg = load * warp_id;
    int end = min(edge_num, beg+ load);
    beg += lane_id;
    int cur_offset = Y[warp_id];
    
    for(int i = beg; i < end; i+=32){
	int mask = __ballot(pred[L[i].srcIndex]);
	int local_id = __popc(mask << (32 - 1) - lane_id) - 1;
	if(pred[L[i].srcIndex]){
	    T[cur_offset+local_id]= L[i];
	}
	cur_offset += __popc(mask);
    }

}

// OUTCORE
void impl2_outcore(vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream& outputFile, bool sortBySource){

	double t_filter, t_comp;
	t_filter = 0;
	t_comp = 0;

	unsigned int *initDist, *distance_cur, *distance_prev, *to_process_arr, *pred; 
	int *anyChange;
	int *hostAnyChange = (int*)malloc(sizeof(int));
	edge_node *edge_list, *L, *T;
	unsigned int edge_num, to_process_num;
	unsigned int *temp = (unsigned int*)malloc(sizeof(unsigned int));
	
	int thread_num = blockSize * blockNum;
	edge_num = count_edges(*graph);
	int warp_num = (thread_num % 32 == 0) ? thread_num/32 : thread_num/32 + 1;
	edge_list = (edge_node*) malloc(sizeof(edge_node)*edge_num);
	initDist = (unsigned int*)calloc(graph->size(),sizeof(unsigned int));	
	pull_distances(initDist, graph->size());
	pull_edges(*graph, edge_list, edge_num);

	if(sortBySource){
	    qsort(edge_list, edge_num, sizeof(edge_node), cmp_edge);
	}

	unsigned int *hostDistanceCur = new unsigned int[graph->size()];
	unsigned int *hostTPA = new unsigned int[warp_num];

	hipMalloc((void**)&distance_cur, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMalloc((void**)&distance_prev, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMalloc((void**)&anyChange, (size_t)sizeof(int));
	hipMalloc((void**)&L, (size_t)sizeof(edge_node)*edge_num);
	hipMalloc((void**)&to_process_arr, (size_t)sizeof(unsigned int)*warp_num);
	hipMalloc((void**)&pred, (size_t)sizeof(unsigned int)*(graph->size()));

	hipMemcpy(distance_cur, initDist, (size_t)sizeof(unsigned int)*(graph->size()), hipMemcpyHostToDevice);
	hipMemcpy(distance_prev, initDist, (size_t)sizeof(unsigned int)*(graph->size()), hipMemcpyHostToDevice);
	hipMemcpy(L, edge_list, (size_t)sizeof(edge_node)*edge_num, hipMemcpyHostToDevice);
	
	hipMemset(anyChange, 0, (size_t)sizeof(int));
	hipMemset(to_process_arr, 0, (size_t)sizeof(unsigned int)*warp_num);
	hipMemset(pred, 0, (size_t)sizeof(unsigned int)*(graph->size()));

	for(int i=0; i < ((int) graph->size())-1; i++){

		setTime();

		if(i == 0){
		    edge_process_outcore<<<blockNum,blockSize>>>(L, edge_num, distance_prev, distance_cur, anyChange, pred);
		    hipDeviceSynchronize();
		} else {
		    hipMemset(pred, 0, (size_t)sizeof(unsigned int)*(graph->size()));
		    edge_process_outcore<<<blockNum,blockSize>>>(T, to_process_num, distance_prev, distance_cur, anyChange, pred);
		    hipDeviceSynchronize();
		    hipFree(T);
		}

		t_comp += getTime();

		hipMemcpy(hostAnyChange, anyChange, sizeof(int), hipMemcpyDeviceToHost);

		if(!hostAnyChange[0]){
			break;
		} else {
			hipMemset(anyChange, 0, (size_t)sizeof(int));
			hipMemcpy(distance_prev, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToDevice);
			hipMemcpy(hostDistanceCur, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToHost);
		}

		if(i == graph->size() - 2){
		    break;
		} else {

		    setTime();

		    hipMemset(to_process_arr, 0, (size_t)sizeof(unsigned int)*warp_num);

		    getX<<<blockNum, blockSize>>>(L, edge_num, pred, to_process_arr);
		    hipDeviceSynchronize();

		    hipMemcpy(temp, to_process_arr + warp_num - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);

		    to_process_num = *temp;

		    hipMemcpy(hostTPA, to_process_arr, sizeof(unsigned int)*warp_num, hipMemcpyDeviceToHost);
		    thrust::exclusive_scan(hostTPA, hostTPA + warp_num, hostTPA);
		    hipMemcpy(to_process_arr, hostTPA, sizeof(unsigned int)*warp_num, hipMemcpyHostToDevice);

		    hipMemcpy(temp, to_process_arr + warp_num - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
		    to_process_num += *temp;

		    hipMalloc((void**)&T, (size_t)sizeof(edge_node)*to_process_num);
		    
		    getT<<<blockNum, blockSize>>>(L, edge_num, pred, to_process_arr, T);
		    hipDeviceSynchronize();

		    t_filter += getTime();
		}
	}

	printf("Computation Time: %f ms\nFiltering Time: %f ms\n", t_comp, t_filter);

	hipMemcpy(hostDistanceCur, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToHost);

	for(int i=0; i < graph->size(); i++){
		if(hostDistanceCur[i] == UINT_MAX){
		    outputFile << i << ":" << "INF" << endl;
		}else{
		    outputFile << i << ":" << hostDistanceCur[i] << endl; 
		}
	}

	hipFree(distance_cur);
	hipFree(distance_prev);
	hipFree(anyChange);
	hipFree(L);
	
	delete[] hostTPA;
	delete[] hostDistanceCur;
	free(initDist);
	free(edge_list);
}


// INCORE
void impl2_incore(vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream& outputFile, bool sortBySource){

	double t_filter, t_comp;
	t_comp = 0;
	t_filter = 0;

	unsigned int *initDist, *distance, *to_process_arr, *pred; 
	int *anyChange;
	int *hostAnyChange = (int*)malloc(sizeof(int));
	edge_node *edge_list, *L, *T;
	unsigned int edge_num, to_process_num;
	unsigned int *temp = (unsigned int*)malloc(sizeof(unsigned int));

	int thread_num = blockSize * blockNum;
	edge_num = count_edges(*graph);
	int warp_num = (thread_num % 32 == 0) ? thread_num/32 : thread_num/32 + 1;
	edge_list = (edge_node*) malloc(sizeof(edge_node)*edge_num);
	initDist = (unsigned int*)calloc(graph->size(),sizeof(unsigned int));	
	pull_distances(initDist, graph->size());
	pull_edges(*graph, edge_list, edge_num);

	unsigned int *hostTPA = new unsigned int[warp_num];

	if(sortBySource){
	    qsort(edge_list, edge_num, sizeof(edge_node), cmp_edge);
	}

	hipMalloc((void**)&distance, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMalloc((void**)&anyChange, (size_t)sizeof(int));
	hipMalloc((void**)&L, (size_t)sizeof(edge_node)*edge_num);
	hipMalloc((void**)&to_process_arr, (size_t)sizeof(unsigned int)*warp_num);
	hipMalloc((void**)&pred, (size_t)sizeof(unsigned int)*(graph->size()));

	hipMemcpy(distance, initDist, (size_t)sizeof(unsigned int)*(graph->size()), hipMemcpyHostToDevice);
	hipMemcpy(L, edge_list, (size_t)sizeof(edge_node)*edge_num, hipMemcpyHostToDevice);
	
	hipMemset(anyChange, 0, (size_t)sizeof(int));
	hipMemset(to_process_arr, 0, (size_t)sizeof(unsigned int)*warp_num);
	hipMemset(pred, 0, (size_t)sizeof(unsigned int)*(graph->size()));

	for(int i=0; i < ((int) graph->size())-1; i++){
		
		setTime();

		if( i == 0 ){
		    edge_process_incore<<<blockNum,blockSize>>>(L, edge_num, distance, anyChange, pred);
		} else {
		    hipMemset(pred, 0, (size_t)sizeof(unsigned int)*(graph->size()));
		    edge_process_incore<<<blockNum, blockSize>>>(T, to_process_num, distance, anyChange, pred);
		    hipFree(T);

		}

		t_comp += getTime();

		hipMemcpy(hostAnyChange, anyChange, sizeof(int), hipMemcpyDeviceToHost);
		if(!hostAnyChange[0]){
		    break;
		} else {
		    hipMemset(anyChange, 0, (size_t)sizeof(int));
		}

		if(i == graph->size() - 2){
		    break;
		} else {

		    setTime();

		    hipMemset(to_process_arr, 0, (size_t)sizeof(unsigned int)*warp_num);

		    getX<<<blockNum, blockSize>>>(L, edge_num, pred, to_process_arr);
		    hipDeviceSynchronize();

		    hipMemcpy(temp, to_process_arr + warp_num - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);

		    to_process_num = *temp;

		    hipMemcpy(hostTPA, to_process_arr, sizeof(unsigned int)*warp_num, hipMemcpyDeviceToHost);
		    thrust::exclusive_scan(hostTPA, hostTPA + warp_num, hostTPA);
		    hipMemcpy(to_process_arr, hostTPA, sizeof(unsigned int)*warp_num, hipMemcpyHostToDevice);
		    hipMemcpy(temp, to_process_arr + warp_num - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);

		    to_process_num += *temp;

		    hipMalloc((void**)&T, (size_t)sizeof(edge_node)*to_process_num);

		    getT<<<blockNum, blockSize>>>(L, edge_num, pred, to_process_arr, T);
		    hipDeviceSynchronize();

		    t_filter += getTime();
		}
	}

	printf("Computation Time: %f ms\nFiltering Time: %f ms\n", t_comp, t_filter);

	unsigned int *hostDistance = (unsigned int *)malloc((sizeof(unsigned int))*(graph->size()));	
	hipMemcpy(hostDistance, distance, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToHost);

	for(int i=0; i < graph->size(); i++){
		if(hostDistance[i] == UINT_MAX){
		    outputFile << i << ":" << "INF" << endl;
		}else{
		    outputFile << i << ":" << hostDistance[i] << endl; 
		}
	}

	hipFree(distance);
	hipFree(anyChange);
	hipFree(L);
	hipFree(to_process_arr);
		        
	delete[] hostTPA;
	delete[] hostDistance;
	free(initDist);
	free(edge_list);
}

